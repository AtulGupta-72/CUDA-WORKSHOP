//PROGRAM1
//THIS PROGRAM DEMONSTRATE VECTOR-MATRIX MULTIPLICATION USING GPU GLOBAL MEMORY
//WITHOUT MEASUING TIME
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

__global__ void VMMulti(float *, float*, float*, int, int);
void VMMultiSerial(float *, float* , float *, int , int );

int main()
{
  int i; //loop variable
  int blockSize=128, blocks; //for cuda blocks
  hipError_t err;//for error checking in cuda API
 // pointers to vector V , matrix M and their result R=V*M on Host/CPU
     float *V, *M, *R; 
 // pointers to vector V, matrix M and for R, on Device/GPU gV and gM
  float * gV, *gM, *gR;
  /***********************************/
  float timespentCPU, timespentGPU;
  clock_t start1, stop1;
  /**********************************/
    hipEvent_t start, stop; 
	hipEventCreate(&start); //Creates an event object 
     hipEventCreate(&stop);
  /*************************************/
/***************************************************************/
 //Define their sizes
  int Vsize=3; //Vector size (1 x Vsize)
  int Mcols=3; //Columns in M (Vsize x Mcols)

 /***************************************************************/
  //Allocate space on Host for V, M, R on CPU
    V=(float*)malloc(Vsize*sizeof(float));
    M=(float*)malloc(Vsize*Mcols*sizeof(float));
	R=(float*)malloc(Mcols*sizeof(float));
  //check the allocations
  if( (V==NULL)||(M==NULL)||(R==NULL))
  {
	  printf("\n Unable to allocate space on CPU for either V/ M/ R ");
	  exit(EXIT_FAILURE);
  }
 /****************************************************************/  
  //Allocate space on Device for gV, gM, gR and check error if any
      err=hipMalloc((void**)&gV,Vsize*sizeof(float));
	   //check the allocation
	     if (hipSuccess!=err)
		 {
			 printf("\n Memory allocation failed on GPU for gV");
			 printf("\n error is- %s", hipGetErrorString(err));
			 exit(EXIT_FAILURE);
		 }
	 err=hipMalloc((void **)&gM,Vsize*Mcols*sizeof(float));
	   //check the allocation
	     if (hipSuccess!=err)
		 {
			 printf("\n Memory allocation failed on GPU for gM");
			 printf("\n error is- %s", hipGetErrorString(err));
			 exit(EXIT_FAILURE);
		 }
	 err=hipMalloc((void **)&gR,Mcols*sizeof(float));
	   //check the allocation
	     if (hipSuccess!=err)
		 {
			 printf("\n Memory allocation failed on GPU for gR");
			 printf("\n error is- %s", hipGetErrorString(err));
			 exit(EXIT_FAILURE);
		 }
  /***********************************************************************/

  //Initialize V and M with random values  
     for(i=0; i<Vsize;i++)
	 {
	    V[i]= (float) (rand()% 10);
		//printf("\n%f",V[i]);
	 }
	 //M is assumed to be stored in column major
     for(i=0; i<(Vsize*Mcols);i++)
	 {
		 M[i]= (float) (rand()% 10);
	    //printf("\n%f", M[i]);
	 }
  /***********************************************************************/
  //Copy V and M  from CPU to GPU
	if (hipSuccess!=hipMemcpy(gV,V,Vsize*sizeof(float),hipMemcpyHostToDevice))
	{
		printf("\n Error in copying V to gV");
		exit(EXIT_FAILURE);
	}
	if (hipSuccess!=hipMemcpy(gM,M,Vsize*Mcols*sizeof(float),hipMemcpyHostToDevice))
    {
		printf("\n Error in copying M to gM");
		exit(EXIT_FAILURE);
	}
 /***********************************************************************/
  //Compute number of cuda blocks needed to compute Mcols
  //elements of R=V*M
	blocks=(int)(Mcols/blockSize);
	if ((Mcols%blockSize)>0)
		blocks++;
	printf("\n The number of blocks needed=%d", blocks);
 /**********************************************************************/
  //Call the cuda kernel VMMUlti for computing R=V*M on GPU
	hipEventRecord(start, 0); //Timestamp, zero default stream
	VMMulti<<<blocks,blockSize>>>(gV, gM,gR,Vsize, Mcols);
	hipDeviceSynchronize(); //synchronize CPU and GPU
	hipEventRecord(stop, 0); //Timestamp
     hipEventSynchronize(stop); 
	hipEventElapsedTime(&timespentGPU, start, stop); 
	printf("\n timespent on GPU=%f",timespentGPU);
	getchar();
/***********************************************************************/
  //call the serial function
	//VMMultiSerial(V, M, R, Vsize, Mcols);
/************************************************************************/
  //Copy Result back to CPU in R
    if (hipSuccess!=hipMemcpy(R,gR,Mcols*sizeof(float),hipMemcpyDeviceToHost))
    {
		printf("\n Error in copying gR to R");
		exit(EXIT_FAILURE);
	}
	/*********************************************************************/
   //Print result on CPU
  	 for(i=0; i<Mcols;i++)
	 {
	    printf("\n%f",R[i]);
	 }
   /*********************************************************************/
   //call the serial function
	 start1=clock();
	 VMMultiSerial(V,M,R,Vsize,Mcols);
	 stop1=clock();
		 timespentCPU = ((float)(stop1 - start1))/CLOCKS_PER_SEC;
		 printf("\n timespent on CPU=%f",timespentCPU);
		 getchar();
		 printf("\n speed up=%f",(float)(timespentCPU/timespentGPU));
	//Do clean up
	    free(V);free(M); free(R); //CPU pointers
	   hipFree(gV); hipFree(gM); hipFree(gR); // Device pointers
	    //Destroy events
	    hipEventDestroy(start); 
        hipEventDestroy(stop); 
        getchar();
   return 0;
}


__global__ void VMMulti(float *gV, float*gM, float *gR, int Vsize, int Mcols)
{
	int i, j;
	float sum=0.0;
	 i=(blockIdx.x*blockDim.x)+threadIdx.x;  //for multi block
	//gM is stored in column major order
	if(i<Mcols)
	{
	  for(j=0;j<Vsize;j++)
		  sum=sum+(gV[j]*gM[j+(i*Vsize)]);
	     __syncthreads();
	    gR[i]=sum;
	}
}


//Serial C function to compute R=V*M
void VMMultiSerial(float *V, float* M, float *R, int Vsize, int Mcols)
{
	int i, j;
	float sum;
	//compute Mcols dot products
	for(i=0;i<Mcols;i++)
	{
		sum=0.0;
	 for(j=0;j<Vsize;j++)
	 {
		 sum=sum+(V[j]*M[j+(i*Vsize)]);

	 }
	    R[i]=sum;
	}
}