
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

/*
	CUDA - 'Hello world' Program. 
	This program adds two Vectors m, n 
	m + n = p 

	Learning objective: Introduction 
	* Creating Blocks
	* Creating Threads
	* Kernel Function
	* cudaMalloc
	* CudaMemcpy
	* Launch Kernel
	* threadIdx
	* cudaFree
*/


__global__ void arradd(int* md, int* nd, int* pd, int size)
{
	int myid = threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int main()
{
	int size = 200 * sizeof(int);
	int m[200], n[200], p[200],*md, *nd,*pd;
	int i=0;

	
	for(i=0; i<200; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}

	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(200, 1);   


	arradd<<< DimGrid,DimBlock >>>(md,nd,pd,size);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	hipFree(md); 
	hipFree(nd);
	hipFree(pd);

	for(i=0; i<200; i++ )
	{
		printf("\t%d",p[i]);
	}	
}




